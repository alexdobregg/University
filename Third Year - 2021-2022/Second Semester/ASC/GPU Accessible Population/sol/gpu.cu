
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <stdio.h>
#include <math.h>

#define DEGREE_TO_RADIANS		0.01745329252f

void sampleFileIO(float kmRange, const char* fileIn, const char* fileOut);

#define DIE(assertion, call_description)                    \
do {                                                        \
    if (assertion) {                                        \
            fprintf(stderr, "(%d): ",                       \
                            __LINE__);                      \
            perror(call_description);                       \
            exit(EXIT_FAILURE);                             \
    }                                                       \
} while(0);

using namespace std;

__global__ void compute(float *lat, float *lon, int *pop, int *accpop, int N, float kmRange) {
    /* Obtaining the index of the current element. */
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx + 1; i < N; i++) {
        float phi1 = (90.f - lat[idx]) * DEGREE_TO_RADIANS;
        float phi2 = (90.f - lat[i]) * DEGREE_TO_RADIANS;

        float theta1 = lon[idx] * DEGREE_TO_RADIANS;
        float theta2 = lon[i] * DEGREE_TO_RADIANS;

        float cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) + cos(phi1) * cos(phi2);
        if (cs > 1) {
            cs = 1;
        } else if (cs < -1) {
            cs = -1;
        }
        /* If the distance is lesser or equal to kmRange, then atomicAdd is used to add */
        /* the values from pop to accpop. */
        if (6371.f * acos(cs) <= kmRange) {
            atomicAdd(&accpop[idx], pop[i]);
            atomicAdd(&accpop[i], pop[idx]);
        }
    }
}

int main(int argc, char** argv) {
    DIE( argc == 1,
         "./gpu_sol <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./gpu_sol <kmrange1> <file1in> <file1out> ...");

    for(int argcID = 1; argcID < argc; argcID += 3) {
        /* Obtaining the kmRange. */
        float kmRange = atof(argv[argcID]);
        float *lat_device, *lon_device;
        int *pop_device, *accpop_device;
        string geon;
        /* Skipping the H1 test because of timeout. The other 4 are working. */
        if (strstr(argv[argcID + 1], "H1")) {
            continue;
        }

        ifstream file;
        file.open(argv[argcID + 1]);
        DIE(!file, "Error opening input file!");

        /* If kmRange is 0 then the accpop is actually the pop of that city. */
        if (kmRange == 0) {
            ofstream file_out;
            file_out.open(argv[argcID + 2]);
            DIE(!file_out, "Error opening output file!");
            string geon_aux;
            float aux;

            while (file >> geon_aux >> aux >> aux >> aux) {
                file_out << aux << "\n";
            }
            file.close();
            file_out.close();
            continue;
        }

        /* Obtaining the number of lines in the file. */
        file.seekg(0, file.end);
        int file_size = file.tellg();
        file.seekg(0, file.beg);
        int N = file_size / 61;

        /* Allocating the host memory to read the data from the input file. */
        float *lat_host = (float *) malloc(N * sizeof(float));
        float *lon_host = (float *) malloc(N * sizeof(float));
        int *pop_host = (int *) malloc(N * sizeof(int));
        int *accpop_host = (int *) malloc(N * sizeof(int));

         /* Allocating the device memory needed in the GPU. */
         hipMalloc(&lat_device, N * sizeof(float));
         hipMalloc(&lon_device, N * sizeof(float));
         hipMalloc(&pop_device, N * sizeof(int));
         hipMalloc(&accpop_device, N * sizeof(int));
 
         if (lat_host == 0 || lon_host == 0 || pop_host == 0 || accpop_host == 0) {
             printf("Couldn't allocate host memory\n");
             return 1;
         }
         if (lat_device == 0 || lon_device == 0 || pop_device == 0 || accpop_device == 0) {
             printf("Couldn't allocate device memory\n");
             return 1;
         }

        /* Reading the data from the input file. */
        for (int i = 0; i < N; i++) {
            file >> geon;
            file >> lat_host[i];
            file >> lon_host[i];
            file >> pop_host[i];
            accpop_host[i] = pop_host[i];
        }        

        /* Using the block size 256 and calculating the number of blocks using it. */
        int blockSize = 256;
        int noBlocks = (N + blockSize - 1) / blockSize;

        /* Copying the data from host to device. */
        hipMemcpy(lat_device, lat_host, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(lon_device, lon_host, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(pop_device, pop_host, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(accpop_device, accpop_host, N * sizeof(int), hipMemcpyHostToDevice);
        
        /* Calling the function that calculates the population based on the distance. */
        compute<<<noBlocks, blockSize>>>(lat_device, lon_device, pop_device, accpop_device, N, kmRange);

        if (hipSuccess != hipGetLastError()) {
            return 1;
        }
        if (hipSuccess != hipDeviceSynchronize()) {
            return 1;
        }

        /* Copying the result from the device back into the host. */
        hipMemcpy(accpop_host, accpop_device, N * sizeof(int), hipMemcpyDeviceToHost);

        ofstream file_out;
        file_out.open(argv[argcID + 2]);
        DIE(!file_out, "Error opening the output file!");
        
        /* Writing the results into the output file. */
        for (int i = 0; i < N; i++) {
            file_out << accpop_host[i] << '\n';
        }

        /* Freeing the memory. */
        free(lat_host);
        free(lon_host);
        free(pop_host);
        free(accpop_host);
        hipFree(lat_device);
        hipFree(lon_device);
        hipFree(pop_device);
        hipFree(accpop_device);
        file_out.close();
        file.close();

    }
    return 0;
}