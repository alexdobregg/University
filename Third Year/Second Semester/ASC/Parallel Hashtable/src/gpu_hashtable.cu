#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

// Function used by murmur_hash
__device__ uint32_t murmur_scramble(uint32_t k) {
	k *= 0xcc9e2d51;
    k = (k << 15) | (k >> 17);
    k *= 0x1b873593;
    return k;
}

/* Function to calculate the hash based on the key using the murmur hash
algorithm, disregarding the len parameter. Also seed is set to a random number
between 2000000000 & 3000000000. Given the fact that rand() is a host function,
a computation would be necessary everytime before calling this function. Therefore,
this is why the seed is hardcoded to 2100682596*/
__device__ uint32_t murmur_hash(int key) {
    uint32_t h = 2100682596;
    uint32_t k = key;
    h ^= murmur_scramble(k);
    h = (h << 13) | (h >> 19);
    h = h * 5 + 0xe6546b64;
    h ^= murmur_scramble(k);
    h ^= h >> 16;
    h *= 0x85ebca6b;
    h ^= h >> 13;
    h *= 0xc2b2ae35;
    h ^= h >> 16;
    return h; 
}


/* The kernel function executed on the GPU to realloc the values from the curr_entries variable
to the new_entries variable which has a bigger capacity. It is based on the linear probing.*/
__global__ void kernel_reshape(Entry *new_entries, int new_max_size, Entry *curr_entries, int curr_max_size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x ;
	if (idx < curr_max_size && curr_entries[idx].key != 0) {
		int key = curr_entries[idx].key;
		int value = curr_entries[idx].value;
		uint32_t hash = murmur_hash(key) % new_max_size;

		while (atomicCAS(&new_entries[hash].key, 0, key) != 0) {
			hash = (hash + 1) % new_max_size;
		}
		new_entries[hash].value = value;
	}
}

/* The kernel function executed on the GPU to insert a new value into the entries array, also based on the linear
probing. If the key already exists, update its value and subtract from the size, otherwise insert a new pair of key-value.*/
__global__ void kernel_insert(Entry *entries, int numKeys, int max_size, int *keys, int *values, int *updated_keys) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x ;
	if (idx < numKeys) {
		int key = keys[idx];
		int value = values[idx];
		uint32_t hash = murmur_hash(key) % max_size;
		int old_key = atomicCAS(&entries[hash].key, 0, key);

		while (old_key != 0 && old_key != key) {
			hash = (hash + 1) % max_size;
			old_key = atomicCAS(&entries[hash].key, 0, key);
		}
		entries[hash].value = value;

		if (old_key == key)
			atomicAdd(updated_keys, 1);
	}
}

/*The kernel function executed on GPU to extract the values and save them in the values variable.
It also searches based on the linear probing*/
__global__ void kernel_get(Entry *entries, int numKeys, int max_size, int *keys, int *values) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x ;
	if (idx < numKeys) {
		int key = keys[idx];
		uint32_t hash = murmur_hash(key) % max_size;

		while (key != entries[hash].key) {
			hash = (hash + 1) % max_size;
		}
		values[idx] = entries[hash].value;
	}
}

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	hashtable.curr_size = 0;
	hashtable.max_size = size;

	glbGpuAllocator->_cudaMalloc((void **)&hashtable.entries, size * sizeof(Entry));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for the hashtable entries!");
		exit(0);
	}

	hipMemset(hashtable.entries, 0, size * sizeof(Entry));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't set the values of entries to 0!");
		exit(0);
	}
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(hashtable.entries);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the hashtable entries");
		exit(0);
	}
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	Entry *new_entries;
	int block_size = 256;
    int blocks_number = (hashtable.max_size + block_size - 1) / block_size;

	glbGpuAllocator->_cudaMalloc((void **)&new_entries, numBucketsReshape * sizeof(Entry));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for the new entries!");
		exit(0);
	}

	hipMemset(new_entries, 0, numBucketsReshape * sizeof(Entry));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't set the values of the new entries to 0!");
		exit(0);
	}

	kernel_reshape<<<blocks_number, block_size>>>(new_entries, numBucketsReshape, hashtable.entries, hashtable.max_size);
	if (hipSuccess != hipDeviceSynchronize()) {
		printf("Couldn't synchronize after kernel reshape!");
		exit(0);
	}

	glbGpuAllocator->_cudaFree(hashtable.entries);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the memory for the current hashtable entries");
		exit(0);
	}

	hashtable.entries = new_entries;
	hashtable.max_size = numBucketsReshape;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int *keys_device;
	int *values_device;
	int *updated_keys;
	int block_size = 256;
    int blocks_number = (numKeys + block_size - 1) / block_size;

	glbGpuAllocator->_cudaMalloc((void **)&keys_device, numKeys * sizeof(int));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for device keys!");
		exit(0);
	}

	glbGpuAllocator->_cudaMalloc((void **)&values_device, numKeys * sizeof(int));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for device values!");
		exit(0);
	}

	glbGpuAllocator->_cudaMallocManaged((void **)&updated_keys, sizeof(int));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for updated_keys variable!");
		exit(0);
	}
	*updated_keys = 0;

	hipMemcpy(keys_device, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't copy the host keys over device keys!");
		exit(0);
	}

	hipMemcpy(values_device, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't copy the host values over device values!");
		exit(0);
	}

	if ((hashtable.curr_size + numKeys) / hashtable.max_size >= LOAD_FACTOR_MAX)
		reshape((hashtable.curr_size + numKeys) / LOAD_FACTOR_MIN);
	

	kernel_insert<<<blocks_number, block_size>>>(hashtable.entries, numKeys, hashtable.max_size, keys_device, values_device, updated_keys);
	if (hipSuccess != hipDeviceSynchronize()) {
		printf("Couldn't synchronize after kernel insert!");
		exit(0);
	}

	hashtable.curr_size = hashtable.curr_size + numKeys - *updated_keys;

	glbGpuAllocator->_cudaFree(keys_device);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the device keys!");
		exit(0);
	}
	
	glbGpuAllocator->_cudaFree(values_device);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the device values!");
		exit(0);
	}

	glbGpuAllocator->_cudaFree(updated_keys);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the updated_keys variable!");
		exit(0);
	}

	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *values_device;
	int *keys_device;
	
	int block_size = 256;
    int blocks_number = (numKeys + block_size - 1) / block_size;

	glbGpuAllocator->_cudaMalloc((void **)&keys_device, numKeys * sizeof(int));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for the device keys!");
		exit(0);
	}

	hipMemcpy(keys_device, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't copy the host keys over device keys!");
		exit(0);
	}

	glbGpuAllocator->_cudaMalloc((void **)&values_device, numKeys * sizeof(int));
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't allocate memory for the device values!");
		exit(0);
	}

	kernel_get<<<blocks_number, block_size>>>(hashtable.entries, numKeys, hashtable.max_size, keys_device, values_device);
	if (hipSuccess != hipDeviceSynchronize()) {
		printf("Couldn't synchronize after kernel get!");
		exit(0);
	}

	int *values_host = (int *) malloc(numKeys * sizeof(int));
	if (!values_host) {
		printf("Couldn't allocate memory for host values");
		exit(0);
	}

	hipMemcpy(values_host, values_device, numKeys * sizeof(int), hipMemcpyDeviceToHost);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't copy the device values over host values!");
		exit(0);
	}

	glbGpuAllocator->_cudaFree(keys_device);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the device keys!");
		exit(0);
	}

	glbGpuAllocator->_cudaFree(values_device);
	if (hipSuccess != hipGetLastError()) {
		printf("Couldn't free the device values!");
		exit(0);
	}

	return values_host;
}
